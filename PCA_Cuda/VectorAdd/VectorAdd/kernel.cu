#include "hip/hip_runtime.h"
#include "kernel.h"
#define dev_dim 400
__global__ void arradd(double* d_m, double* d_n, double size, double projectionValue) {
	int myid = threadIdx.x;

	d_n[myid] = d_n[myid] + projectionValue * d_m[myid];
}
__global__ void arradd2(double* d_m, double* d_n, double size) {
	int myid = threadIdx.x;

	d_n[myid] += d_m[myid];
}

void cudaAdd(double m[dev_dim], double n[dev_dim], double size, double projectionValue) {

	double *d_m, *d_n;

	hipMalloc(&d_m, size);
	hipMalloc(&d_n, size);

	hipMemcpy(d_m, m, size, hipMemcpyHostToDevice);

	dim3 DimGrid(1, 1);
	dim3 DimBlock(dev_dim, 1);

	arradd <<< DimGrid, DimBlock >>> (d_m, d_n, size, projectionValue);
	// arradd <<< 1, 200 >>> (d_m, d_n, d_p, size);

	hipMemcpy(n, d_n, size, hipMemcpyDeviceToHost);

	hipFree(d_m);
	hipFree(d_n);
}

void cudaAdd2(double m[dev_dim], double n[dev_dim], double size) {

	double *d_m, *d_n;

	hipMalloc(&d_m, size);
	hipMalloc(&d_n, size);

	hipMemcpy(d_m, m, size, hipMemcpyHostToDevice);

	dim3 DimGrid(1, 1);
	dim3 DimBlock(dev_dim, 1);

	arradd2 << < DimGrid, DimBlock >> > (d_m, d_n, size);
	// arradd <<< 1, 200 >>> (d_m, d_n, d_p, size);

	hipMemcpy(n, d_n, size, hipMemcpyDeviceToHost);

	hipFree(d_m);
	hipFree(d_n);
}



host_vector<Vmatrix> cudaReconstruction(vector<Arr_dim> cuda_means, Vmatrix cuda_oneImage_scores, vector<vector<Arr_dim>> cuda_oneVecs, int cellImage_num, int dim, int pca_dim) {
	cout << "Start cuda now" << endl;
	host_vector<Vmatrix> whole_OriginalData;
	double size = dev_dim * sizeof(double);
	double *d_m, *d_n;

	hipMalloc(&d_m, size);
	hipMalloc(&d_n, size);
	dim3 DimGrid(1, 1);
	dim3 DimBlock(dev_dim, 1);

	for (int cellIndex = 0; cellIndex < cellImage_num; cellIndex++) {
		Vmatrix OriginalData(1, dim);

		double d_OriginalData[dev_dim];
		for (int i = 0; i < dim; i++) {
			d_OriginalData[i] = 0;
		}

		hipMemcpy(d_n, d_OriginalData, size, hipMemcpyHostToDevice);
		for (int i = 0; i < pca_dim; i++) {
			double projectionValue = cuda_oneImage_scores.data[cellIndex][i];
			
			hipMemcpy(d_m, cuda_oneVecs[cellIndex][i].data, size, hipMemcpyHostToDevice);
			
			arradd <<< DimGrid, DimBlock >>> (d_m, d_n, size, projectionValue);
		}

		hipMemcpy(d_m, cuda_means[cellIndex].data, size, hipMemcpyHostToDevice);
		arradd2 << < DimGrid, DimBlock >> > (d_m, d_n, size);
		hipMemcpy(d_OriginalData, d_n, size, hipMemcpyDeviceToHost);

		for (int i = 0; i < dim; i++) {
			OriginalData.data[0][i] = d_OriginalData[i];
		}
		whole_OriginalData.push_back(OriginalData);

	}

	hipFree(d_m);
	hipFree(d_n);

	return whole_OriginalData;
}





















