#include "hip/hip_runtime.h"
//#include <iostream>
//#include <thrust/host_vector.h> 
//#include <thrust/device_vector.h>
//
//using namespace std;
//using namespace thrust;
//
//#define dev_dim 20
//__global__ void arradd(double* d_m, double* d_n, double size, double projectionValue) {
//	int myid = threadIdx.x;
//
//	d_n[myid] += projectionValue * d_m[myid];
//}
//
//void cudaAdd(double m[dev_dim], double n[dev_dim], double size, double projectionValue) {
//
//	double *d_m, *d_n;
//
//	hipMalloc(&d_m, size);
//	hipMalloc(&d_n, size);
//
//	hipMemcpy(d_m, m, size, hipMemcpyHostToDevice);
//
//	dim3 DimGrid(1, 1);
//	dim3 DimBlock(dev_dim, 1);
//
//	arradd << < DimGrid, DimBlock >> > (d_m, d_n, size, projectionValue);
//	// arradd <<< 1, 200 >>> (d_m, d_n, d_p, size);
//
//	hipMemcpy(n, d_n, size, hipMemcpyDeviceToHost);
//
//	hipFree(d_m);
//	hipFree(d_n);
//}
//
//int main() {
//
//	vector<vector<double>> data1;
//	vector<double> data11;
//	for (int i = 0; i < 20; i++) {
//		data11.push_back(i);
//	}
//	data1.push_back(data11);
//	data1.push_back(data11);
//
//	host_vector<double> H(data11.begin(), data11.end());
//	device_vector<double> D = H;
//	
//	// extract raw pointer from device_ptr
//	//double * raw_ptr = thrust::raw_pointer_cast(D);
//	size_t N = 10;
//	// create a device_ptr
//	thrust::device_ptr<int> dev_ptr = thrust::device_malloc<int>(N);
//	// extract raw pointer from device_ptr
//	int * raw_ptr = thrust::raw_pointer_cast(dev_ptr);
//	//cout << raw_ptr[10] << endl;
//	raw_ptr[10] = 100;
//	cout << D[10] << endl;
//
//
//	double projectionValue = 0.3;
//	double a[dev_dim], b[dev_dim];
//	for (int j = 0; j < 20; j++) {
//		a[j] = j;// oneVec.data[j][i];
//		b[j] = 0; // OriginalData.data[0][j];
//	}
//	double size = dev_dim * sizeof(double);
//	cudaAdd(a, b, size, projectionValue);
//
//	for (int i = 0; i < 20; i++) {
//		cout << b[i] << " ";
//	}
//	cout << endl;
//
//	int x;
//	cin >> x;
//	return 0;
//}


//#include <thrust/device_vector.h>
//#include <thrust/transform.h>
//#include <thrust/functional.h>
//#include <thrust/copy.h>
//#include <iostream>
//
//using namespace std;
//using namespace thrust;
//using namespace thrust::placeholders;
//
//// SAXPY
//struct saxpy {
//	double a;
//	saxpy (double a) : a(a){}
//	__host__ __device__ 
//		double operator()(double x) {
//		return a*x;
//	}
//};
//
//int main(void)
//{
//	// Vector addition Z = X + Y
//	thrust::device_vector<float> X(3);
//	thrust::device_vector<float> Y(3);
//	thrust::device_vector<float> Z(3);
//	X[0] = 10; X[1] = 20; X[2] = 30;
//	Y[0] = 15; Y[1] = 35; Y[2] = 10;
//	thrust::transform(X.begin(), X.end(),
//		Y.begin(),
//		Z.begin(),
//		thrust::plus<float>());
//	for (size_t i = 0; i < Z.size(); i++)
//		std::cout << "Z[" << i << "] = " << Z[i] << "\n";
//
//	// Sum of a vector   result = sum(M)
//	vector<double> test;
//	test.push_back(2);
//	test.push_back(1);
//	test.push_back(8);
//	device_vector<double> M = test;
//	//M[0] = 2; M[1] = 1; M[2] = 10;
//	double result = reduce(M.begin(), M.end());
//	cout << "sum is " << result << endl;
//
//	// SAXPY   N = a * M
//	device_vector<double> N(3);
//	double a = 3;
//	transform(M.begin(), M.end(),
//		N.begin(),
//		saxpy(a));
//	for (size_t i = 0; i < N.size(); i++) {
//		cout << "N[" << i << "] = " << N[i] <<" ";
//	}
//	cout << endl;
//	
//	for (int i = 0; i < 3; i++) {
//		transform(M.begin(), M.end(),
//			N.begin(),
//			N.begin(),
//			_1 + _2);
//	}
//	
//
//	
//	for (size_t i = 0; i < N.size(); i++) {
//		cout << "N[" << i << "] = " << N[i] << " ";
//	}
//	cout << endl;
//	host_vector<double> tmp = N;
//	thrust:: copy(N.begin(), N.end(), test.begin());
//	for (size_t i = 0; i < N.size(); i++) {
//		cout << "N[" << i << "] = " << test[i] << " ";
//	}
//	
//	int x;
//	cin >> x;
//	return 0;
//}





//#include <thrust/device_vector.h>
//
//thrust::device_vector<int> iVec;
//
//int* iArray = thrust::raw_pointer_cast(&iVec[0]);
//
//fooKernel << <x, y >> >(iArray);
//
//// Template structure to pass to kernel
//template <typename T>
//struct KernelArray
//{
//	T*  _array;
//	int _size;
//};
//
//// Function to convert device_vector to structure
//template <typename T>
//KernelArray<T> convertToKernel(thrust::device_vector<T>& dVec)
//{
//	KernelArray<T> kArray;
//	kArray._array = thrust::raw_pointer_cast(&dVec[0]);
//	kArray._size = (int)dVec.size();
//
//	return kArray;
//}
//
//thrust::device_vector<int> iVec;
//
//fooKernel << <x, y >> >(convertToKernel(iVec)); // Explicit conversion from iVec to KernelArray<int>
//
//__global__ fooKernel(KernelArray<int> inArray)
//{
//	for (int i = 0; i < inArray._size; ++i)
//		something = inArray._array[i];
//	// ...
//	return;
//}

#include <iostream>
#include <vector>
using namespace std;
#define dev_dim 10

typedef double arr_dim[dev_dim];

__global__ void arradd(double* d_m, double* d_n, double size) {
	int myid = threadIdx.x;

	d_n[myid] = d_n[myid] + d_m[myid];
}

int main() {
	//arr_dim Ta;
	//arr_dim Tb;
	//double Ta[dev_dim];
	//double Tb[dev_dim];

	//for (int i = 0; i < dev_dim; i++) {
	//	Ta[i] = 0;
	//	Tb[i] = i + 1;
	//}

	// Test
	vector<vector<double *>> test1, test2;
	vector<double *> test11, test22;
	double test111[dev_dim], test222[dev_dim];
	for (int i = 0; i < dev_dim; i++) {
		test111[i] = i;
		test222[i] = 0;
	}
	test11.push_back(test111);
	test22.push_back(test222);
	test1.push_back(test11);
	test2.push_back(test22);


	double *d_m, *d_n;
	double size = dev_dim * sizeof(double);

	hipMalloc(&d_m, size);
	hipMalloc(&d_n, size);

	hipMemcpy(d_n, test2[0][0], size, hipMemcpyHostToDevice);
	hipMemcpy(d_m, test1[0][0], size, hipMemcpyHostToDevice);

	dim3 DimGrid(1, 1);
	dim3 DimBlock(dev_dim, 1);
	arradd << <DimGrid, DimBlock >> > (d_m, d_n, size);
	hipMemcpy(test2[0][0], d_n, size, hipMemcpyDeviceToHost);

	//for (int i = 0; i < dev_dim; i++) {
		//cout << Ta[i] << " ";
	//}
	//cout << endl;

	hipFree(d_m);
	hipFree(d_n);

	
	

	for (int i = 0; i < dev_dim; i++) {
		test1[0][0][i] += 5;
		cout << test1[0][0][i] << " ";
	}
	cout << endl;

	// Copy
	test2 = test1;
	for (int i = 0; i < dev_dim; i++) {
		cout << test2[0][0][i] << " ";
	}
	cout << endl;
	

	int x;
	cin >> x;
	return 0;
}

